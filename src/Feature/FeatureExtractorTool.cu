#include "hip/hip_runtime.h"

#include "FeatureExtractorTool.h"

__global__
void windowFFT_cu(cp *d_SpeechSignal, int frameNum, int frameSize, int f, int selIdx, double arg){
    extern __shared__ char s_SpeechSignal[];
    int p, i, j, rollIdx=0, oldRollIdx;
    size_t innerIdx = threadIdx.x % frameSize, 
           frame_offset = blockDim.x*blockIdx.x+(threadIdx.x/frameSize)*frameSize;
    double temp_cp[2], temp_wm[2], temp_w[2];
    cp *temp = (cp *) temp_cp, 
       *wm = (cp*)temp_wm, 
       *w = (cp*)temp_w; 
    //cp *d_signal[2];
    cp *s_signal[2]; 
    
    size_t sharedSize = blockDim.x * sizeof(cp);
    s_signal[0] = (cp *)s_SpeechSignal;
    s_signal[1] = (cp *)&s_SpeechSignal[sharedSize];
    //d_signal[0] = d_SpeechSignal+frame_offset;
    //d_signal[1] = d_signal[0]+frameNum*frameSize;
    
    *(s_signal[0]+innerIdx) = *(d_SpeechSignal+frame_offset+innerIdx);
    __syncthreads();
    
    for(int k = frameSize>>1; k; k>>=1, arg*=0.5){
        rollIdx ^= 1;
        oldRollIdx = rollIdx^1;
        
        getPolarValue(1, f*arg, temp_wm);
        *temp_w = 1;
        *(temp_w+1) = 0;
        
        i = innerIdx/k;
        j = innerIdx%k;
        for(int t=0; t<i; t++){
            //w = w*wm;
            mulComplex(w,wm,w);
        }
        i = i*k;
        p = i<<1;
        if(p>=frameSize) p-=frameSize;
    
        //mulComplex(temp, w, d_signal[oldRollIdx]+(p+k+j)); 
        //addComplex(d_signal[rollIdx]+(i+j), temp, d_signal[oldRollIdx]+(p+j));
        
        mulComplex(temp, w, s_signal[oldRollIdx]+(p+k+j)); 
        addComplex(s_signal[rollIdx]+(i+j), temp, s_signal[oldRollIdx]+(p+j));
        __syncthreads();
    }
    d_SpeechSignal[frame_offset+innerIdx] = *(s_signal[selIdx]+innerIdx);
}

__global__ 
void fft_cu_part(cp *d_SpeechSignal, int n, int f, double arg){
    int p, i, j, idx, rollIdx=0, oldRollIdx;
    cp* d_signal[2]; 
    d_signal[0] = d_SpeechSignal;
    d_signal[1] = &d_SpeechSignal[n];
    
    int *finalRollIdx = (int *) &d_SpeechSignal[2*n];
    
    idx = blockDim.x*blockIdx.x + threadIdx.x;
    
    //double arg = pi;
    double temp_cp[2], temp_wm[2], temp_w[2];
    cp *temp = (cp *) temp_cp, *wm = (cp*)temp_wm, *w = (cp*)temp_w;
    for(int k = n>>1; k; k>>=1, arg*=0.5){
        rollIdx ^= 1;
        oldRollIdx = rollIdx^1;
        
        //cp wm = std::polar(1.0,f*arg), w(1,0);
        getPolarValue(1, f*arg, temp_wm);
        *temp_w = 1;
        *(temp_w+1) = 0;
        
        i = idx/k;
        j = idx%k;
        for(int t=0; t<i; t++){
            //w = w*wm;
            mulComplex(w,wm,w);
        }
        i = i*k;
        p = i<<1;
        if(p>=n) p-=n;
    
        //d_signal[rollIdx][i+j] = d_signal[oldRollIdx][p+j] + w*d_signal[oldRollIdx][p+k+j];
        mulComplex(temp, w, &d_signal[oldRollIdx][p+k+j]); 
        addComplex(&d_signal[rollIdx][i+j], temp, &d_signal[oldRollIdx][p+j]);
        __syncthreads();
    }
    if(idx==0)
        *finalRollIdx = rollIdx;
}

__device__ 
void mulComplex(cp *output, cp *input1, cp *input2){
    double real1, imag1, real2, imag2;
    getRealImag(real1,imag1,input1);
    getRealImag(real2,imag2,input2);
    double *ptr_output = (double *)output;
    *ptr_output = real1*real2-imag1*imag2;
    *(ptr_output+1) = real1*imag2+imag1*real2;
    //output = cp( real1*real2-imag1*imag2 , real1*imag2+imag1*real2 );
}

__device__
void addComplex(cp *output, cp *input1, cp *input2){
    double real1, imag1, real2, imag2;
    getRealImag(real1,imag1,input1);
    getRealImag(real2,imag2,input2);
    double *ptr_output = (double *)output;
    *ptr_output = real1+real2;
    *(ptr_output+1) = imag1+imag2;
    //output = cp( real1+real2, imag1+imag2 );
}

__device__
void getRealImag(double& real, double& imag, const cp *input){
    double *comp = (double *)input;
    real = *comp;
    imag = *(comp+1);
}

__device__
void getPolarValue(double rho, double theta, double *output){
    *output = rho*cos(theta);
    *(output+1) = rho*sin(theta);
}

